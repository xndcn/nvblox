#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/map/blox.h"
#include "nvblox/map/common_names.h"

namespace nvblox {

// Must be called with:
// - a single block
// - one thread per voxel
__global__ void setColorBlockGray(ColorBlock* block_device_ptr) {
  ColorVoxel* voxel_ptr =
      &block_device_ptr->voxels[threadIdx.z][threadIdx.y][threadIdx.x];
  // NOTE(dtingdahl): This is identical to the CPU initialization defined in
  // voxels.h
  voxel_ptr->color.r = 127;
  voxel_ptr->color.g = 127;
  voxel_ptr->color.b = 127;
  voxel_ptr->color.a = 255;
  voxel_ptr->weight = 0.0f;
}

void setColorBlockGrayOnGPUAsync(ColorBlock* block_device_ptr,
                                 const CudaStream& hip_stream) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  setColorBlockGray<<<1, kThreadsPerBlock, 0, hip_stream>>>(block_device_ptr);
  checkCudaErrors(hipPeekAtLastError());
}

template <class BlockType>
__global__ void initializeBlocksKernel(BlockType** block_ptrs, int num_blocks) {
  const int block_idx = blockIdx.x;

  if (block_idx < num_blocks) {
    block_ptrs[block_idx]->voxels[threadIdx.z][threadIdx.y][threadIdx.x] =
        BlockType::VoxelType();
  }
}

template <class BlockType>
void initializeBlocksAsync(host_vector<BlockType*>& blocks,
                           const CudaStream& hip_stream,
                           const MemoryType /*unused*/) {
  if (blocks.empty()) {
    return;
  }

  const dim3 threads_per_block = {BlockType::kVoxelsPerSide,
                                  BlockType::kVoxelsPerSide,
                                  BlockType::kVoxelsPerSide};
  const int num_blocks = blocks.size();

  initializeBlocksKernel<BlockType>
      <<<num_blocks, threads_per_block, 0, hip_stream>>>(blocks.data(),
                                                          blocks.size());
}

// Specialization for meshblock
template <>
void initializeBlocksAsync(host_vector<MeshBlock*>& blocks,
                           const CudaStream& hip_stream,
                           const MemoryType memory_type) {
  for (auto& ptr : blocks) {
    MeshBlock::initAsync(ptr, memory_type, hip_stream);
  }
}

// Specializations for Voxelblock types
template void initializeBlocksAsync<TsdfBlock>(host_vector<TsdfBlock*>& blocks,
                                               const CudaStream& hip_stream,
                                               const MemoryType memory_type);
template void initializeBlocksAsync<OccupancyBlock>(
    host_vector<OccupancyBlock*>& blocks, const CudaStream& hip_stream,
    const MemoryType memory_type);
template void initializeBlocksAsync<ColorBlock>(
    host_vector<ColorBlock*>& blocks, const CudaStream& hip_stream,
    const MemoryType memory_type);
template void initializeBlocksAsync<MeshBlock>(host_vector<MeshBlock*>& blocks,
                                               const CudaStream& hip_stream,
                                               const MemoryType memory_type);
template void initializeBlocksAsync<FreespaceBlock>(
    host_vector<FreespaceBlock*>& blocks, const CudaStream& hip_stream,
    const MemoryType memory_type);
template void initializeBlocksAsync<EsdfBlock>(host_vector<EsdfBlock*>& blocks,
                                               const CudaStream& hip_stream,
                                               const MemoryType memory_type);

}  // namespace nvblox
